#include "hip/hip_runtime.h"
#include "commons.h"
#include "average_gpu.h"


// STD includes
#include <assert.h>
#include <stdio.h>

// CUDA runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#define BLOCKSIZE_X 16
#define BLOCKSIZE_Y 16

// Utilities and system includes
#include <hip/hip_runtime_api.h>

#define gpuErrChk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, char *file, int line, bool abort=true) {
	if (code != hipSuccess) {
		fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}


static // Print device properties
void printDevProp(hipDeviceProp_t devProp) {
	printf("Major revision number:         %d\n", devProp.major);
	printf("Minor revision number:         %d\n", devProp.minor);
	printf("Name:                          %s\n", devProp.name);
	printf("Total global memory:           %zu\n", devProp.totalGlobalMem);
	printf("Total shared memory per block: %zu\n", devProp.sharedMemPerBlock);
	printf("Total registers per block:     %d\n", devProp.regsPerBlock);
	printf("Warp size:                     %d\n", devProp.warpSize);
	printf("Maximum memory pitch:          %zu\n", devProp.memPitch);
	printf("Maximum threads per block:     %d\n", devProp.maxThreadsPerBlock);
	for (int i = 0; i < 3; ++i)
		printf("Maximum dimension %d of block:  %d\n", i, devProp.maxThreadsDim[i]);
	for (int i = 0; i < 3; ++i)
		printf("Maximum dimension %d of grid:   %d\n", i, devProp.maxGridSize[i]);
	printf("Clock rate:                    %d\n", devProp.clockRate);
	printf("Total constant memory:         %zu\n", devProp.totalConstMem);
	printf("Texture alignment:             %zu\n", devProp.textureAlignment);
	printf("Concurrent copy and execution: %s\n", (devProp.deviceOverlap ? "Yes" : "No"));
	printf("Number of multiprocessors:     %d\n", devProp.multiProcessorCount);
	printf("Kernel execution timeout:      %s\n", (devProp.kernelExecTimeoutEnabled ? "Yes" : "No"));
	return;
}

void print_gpuInfo() {
	int rtVersion = 0;
	printf("*********************************************************************************************\n");
	checkCudaErrors(hipRuntimeGetVersion(&rtVersion));
	printf("CUDA Runtime Version = %d\n", rtVersion);
	int driverVersion = 0;
	checkCudaErrors(hipDriverGetVersion(&driverVersion));
	printf("CUDA Driver Version  = %d\n", rtVersion);

	int numDevices = 0;
	checkCudaErrors(hipGetDeviceCount(&numDevices));
	printf("Devices found        = %d\n", numDevices);

	for (int i = 0; i < numDevices; i++) {
		hipDeviceProp_t properties;
		checkCudaErrors(hipGetDeviceProperties(&properties, i));
		printDevProp(properties);
	}
	printf("*********************************************************************************************\n");
}


/******************************************************************************
* UTILITY FUNCTIONS
******************************************************************************/

__global__ static void averageKernel(
		unsigned char* output,
		unsigned char* input,
		int imageW,
		int imageH
		){

	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	const int posIn = y * imageW + x;
	const int posOut = y * imageW + x;

	unsigned int sum = 0;
	unsigned int numels = 0;
	for (int kY = -KERNEL_RADIUS; kY <= KERNEL_RADIUS; kY++) {
		const int curY = y + kY;
		if (curY < 0 || curY > imageH) {
			continue;
		}

		for (int kX = -KERNEL_RADIUS; kX <= KERNEL_RADIUS; kX++) {
			const int curX = x + kX;
			if (curX < 0 || curX > imageW) {
				continue;
			}

			const int curPosition = (curY * imageW + curX);
			if (curPosition >= 0 && curPosition < (imageW * imageH)) {
				sum += input[curPosition];
				numels++;
			}
		}
	}
	output[y * imageW + x] = (unsigned char) (sum / numels);

	//output[posOut] = input[posIn];

}

__global__ static void readChannel(
		unsigned char *output,
		unsigned char *input,
		int imageW,
		int imageH,
		int channelToExtract,
		int numChannels) {

	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	const int posIn = y * imageW * numChannels + x * numChannels + channelToExtract;
	const int posOut = y * imageW + x;

	output[posOut] = input[posIn];
}

__global__ static void writeChannel(
		unsigned char *output,
		unsigned char *input,
		int imageW,
		int imageH,
		int channelToExtract,
		int numChannels) {

	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	const int posIn = y * imageW + x;
	const int posOut = y * imageW * numChannels + x * numChannels + channelToExtract;

	output[posOut] = input[posIn];
}

/******************************************************************************
* AVERAGE FILTER
******************************************************************************/

void average_gpu(
		unsigned char *inputImage,
		unsigned char *outputImage,
		int imageW,
		int imageH,
		int numChannels
) {
	unsigned char *gpuImage;
	size_t size = imageW * imageH * numChannels * sizeof(unsigned char);

	hipMalloc(&gpuImage, size);
	hipMemcpy(gpuImage, inputImage, size, hipMemcpyHostToDevice);

	dim3 block(BLOCKSIZE_X, BLOCKSIZE_Y);
	dim3 grid(static_cast<unsigned int>(imageW / BLOCKSIZE_X), static_cast<unsigned int>(imageH / BLOCKSIZE_Y));

	unsigned char *channelData;
	unsigned char *averageChannel;
	int nrChannels = 3;
	int ch_size = imageW * imageH;

	for(int i=0; i<numChannels; i++) {
		hipMalloc(&channelData, sizeof(unsigned char) * imageW * imageH);
		hipMalloc(&averageChannel, sizeof(unsigned char) * imageW * imageH);

		readChannel <<< grid, block >>> (channelData, gpuImage, imageW, imageH, i, nrChannels);
		gpuErrChk(hipPeekAtLastError());
		gpuErrChk(hipDeviceSynchronize());

		averageKernel <<< grid, block >>> (averageChannel, channelData, imageW, imageH);
		writeChannel <<< grid, block >>> (gpuImage, averageChannel, imageW, imageH, i, nrChannels);
	}
	hipMemcpy(outputImage, gpuImage, size, hipMemcpyDeviceToHost);
	hipFree(channelData);
	hipFree(averageChannel);
}

